#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <time.h>

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__device__ void ChangeState(int result, int& yes, int& no, int& unknown)
{
	switch (result)
	{
	case 0:
		// both the same
		break;
	case 1:
		// yes + no
		no--;
		yes--;
		unknown += 2;
		break;
	case 2:
		// yes + unknown
		yes++;
		unknown--;
		break;
	case 3:
		// no + unknown
		no++;
		unknown--;
		break;
	default:
		break;
	}
}

int CountSize(int n)
{
	int size = 0;
	for (int i = 1; i <= n + 1; i++)
	{
		size += i;
	}
	return size;
}

__device__ int EvaluateCase(int yes, int no, int unknown, int n, hiprandState* randState)
{
	int randomValue = (int)(hiprand_uniform(&randState[0]) * n) % n + 1;
	// 0 yes, 1 no, 2 unknown
	int firstSelection;
	int secondSelection;
	if (randomValue <= yes)
	{
		firstSelection = 0;
		yes--;
	}
	else if (randomValue <= yes + no)
	{
		firstSelection = 1;
		no--;
	}
	else
	{
		firstSelection = 2;
		unknown--;
	}

	n--;
	randomValue = (int)(hiprand_uniform(&randState[0]) * n) % n + 1;
	if (randomValue <= yes)
	{
		secondSelection = 0;
		yes--;
	}
	else if (randomValue <= yes + no)
	{
		secondSelection = 1;
		no--;
	}
	else
	{
		secondSelection = 2;
		unknown--;
	}
	int result = firstSelection ^ secondSelection;
	return result;
}

__device__ void GetYesNoFromIndex(int index, int n, int& yes, int& no)
{
	int currentIndex = n;
	int iterator = 1;
	if (index <= n)
	{
		yes = 0;
		no = index;
		return;
	}

	while ((currentIndex + n + 1 - iterator) < index)
	{
		currentIndex += n + 1 - iterator;
		iterator++;
	}
	yes = iterator;
	no = index - currentIndex - 1;
}

__device__ int ReturnIndex(int yes, int no, int n)
{
	int index = 0;

	for (int i = 1; i <= yes; i++)
	{
		index += n + 2 - i;
	}

	index += no;
	return index;
}

extern __shared__ int shared[];
__global__ void MonteCarlo(int* outNumerators, int* outDenominators, int states, int peoples, int iterationsNum, int iterationsPerThread)
{
	int* numerator = (int*)shared;
	int* denominator = (int*)& shared[states];
	bool* casesUsed = new bool[states]();
	int tId = threadIdx.x + (blockIdx.x * blockDim.x);
	hiprandState randState;
	hiprand_init((unsigned long long)clock() + tId, 0, 0, &randState);

	// Set shared memory to 0
	for (int i = threadIdx.x; i < states; i += blockDim.x)
	{
		numerator[i] = 0;
		denominator[i] = 0;
	}

	__syncthreads();

	for (int i = 0; i < iterationsNum; i++)
	{
		/*if (iterationsNum <= iterationsPerThread * tId + i)
		{
			break;
		}*/
		bool end = false;
		bool isYesResult = false;
		int test = (int)(hiprand_uniform(&randState) * states) % states;
		int yes = peoples / 2, no = peoples / 2, unknown = peoples - yes - no;
		GetYesNoFromIndex(test, peoples, yes, no);
		unknown = peoples - yes - no;
		for (int j = 0; j < states; j++)
		{
			casesUsed[j] = false;
		}
		casesUsed[ReturnIndex(yes, no, peoples)] = true;
		while (!end)
		{
			int result = EvaluateCase(yes, no, unknown, peoples, &randState);
			ChangeState(result, yes, no, unknown);

			int index = ReturnIndex(yes, no, peoples);
			casesUsed[index] = true;

			if (yes == 0 || no == 0)
			{
				if (yes > 0)
				{
					isYesResult = true;
				}
				end = true;
			}
		}
		for (int j = tId; j < states; j+=blockDim.x)
		{
			if (casesUsed[j])
			{
				if (isYesResult)
				{
					numerator[j]++;
				}
				denominator[j]++;
			}
		}
	}
	delete casesUsed;
	__syncthreads();
	for (int i = threadIdx.x; i < states; i += blockDim.x)
	{
		outNumerators[i] = numerator[i];
		outDenominators[i] = denominator[i];
	}
}

__global__ void CalculateProbabilities(int* numerators, int* denominators, double* winProbabilities, int states)
{
	int tId = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tId < states)
	{
		if (denominators[tId] != 0)
			winProbabilities[tId] = numerators[tId] * 1.0 / denominators[tId];
		else
			winProbabilities[tId] = 0;
	}
}


int main()
{
	// Monte carlo init numbers
	const int iterationsNumber = 1000000;
	const int peoplesNumber = 5;
	const int statesNumber = CountSize(peoplesNumber);
	double* winProbabilities = new double[statesNumber];
	// Threads and blocks
	const int threadsNumber = 256;
	int blocksNumber = 1;
	// Cuda variables
	double* cu_winProbabilities;
	int* cu_statesNumerators;
	int* cu_statesDenominators;
	int simulationsPerThread = ceil(iterationsNumber * 1.0 / threadsNumber);
	hipError_t cudaStatus;

	// Allocate memory on gpu
	cudaStatus = hipMalloc((void**)& cu_winProbabilities, statesNumber * sizeof(double));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate (error code %s)!\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}
	cudaStatus = hipMalloc((void**)& cu_statesNumerators, statesNumber * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate (error code %s)!\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}
	cudaStatus = hipMalloc((void**)& cu_statesDenominators, statesNumber * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate (error code %s)!\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}

	// Set numerators and denominators values to 0
	hipMemset(cu_statesNumerators, 0, statesNumber * sizeof(int));
	hipMemset(cu_statesDenominators, 0, statesNumber * sizeof(int));

	// Monte Carlo simulation
	MonteCarlo << <blocksNumber, threadsNumber, 2 * statesNumber * sizeof(int) >> > (cu_statesNumerators, cu_statesDenominators, statesNumber, peoplesNumber, iterationsNumber, simulationsPerThread);

	// Calculate states probabilities to win
	blocksNumber = (statesNumber + threadsNumber - 1) / threadsNumber;
	CalculateProbabilities << <blocksNumber, threadsNumber >> > (cu_statesNumerators, cu_statesDenominators, cu_winProbabilities, statesNumber);

	// Copy results to host
	hipMemcpy(winProbabilities, cu_winProbabilities, statesNumber * sizeof(double), hipMemcpyDeviceToHost);

	for (int i = 0; i < statesNumber; i++)
	{
		printf("%.5f\n", winProbabilities[i]);
	}

	// Free memory
	hipFree(cu_winProbabilities);
	hipFree(cu_statesNumerators);
	hipFree(cu_statesDenominators);

	delete winProbabilities;

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)& dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)& dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)& dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> > (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
